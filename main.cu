#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include<time.h>
#include "vec3.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(color* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    fb[pixel_index] = color(float(i) / max_x, float(j) / max_y, 0.2f);
}

int main() {
    int image_width = 1200;
    int image_height = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(color);

    // allocate FB
    color* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, cuda_stop, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render << <blocks, threads >> > (fb, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    cuda_stop = clock();
    double timer_seconds = ((double)(cuda_stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Cuda computation took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    uint8_t* pixels = new uint8_t[image_width * image_height * 3];
    int write_index = 0;
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            color pixelColor = fb[pixel_index];
            int ir = int(255.99 * pixelColor.x());
            int ig = int(255.99 * pixelColor.y());
            int ib = int(255.99 * pixelColor.z());
            pixels[write_index++] = ir;
            pixels[write_index++] = ig;
            pixels[write_index++] = ib;
        }
    }

    stbi_write_bmp("image.bmp", image_width, image_height, 3, pixels);
    stop = clock();
    timer_seconds = ((double)(stop - cuda_stop)) / CLOCKS_PER_SEC;
    std::cerr << "Image write took " << timer_seconds << " seconds. \n";
 
    checkCudaErrors(hipFree(fb));


}