#include "hip/hip_runtime.h"
﻿
#include "commons.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include<time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include "sphere.h"
#include "material.h"
#include "camera.h"
#include "hittable_list.h"
#include "scene.h"
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define TBP 512;


__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984+ pixel_index, 0, 0, &rand_state[pixel_index]);
}

__device__ color ray_color(const ray& r, const color& background, const hittable** world, hiprandState* local_rand_state,  int depth) {
    hit_record rec;

    // If we've exceeded the ray bounce limit, no more light is gathered.
    if (depth <= 0)
        return color(0, 0, 0);

    // If the ray hits nothing, return the background color.
    if (!(*world)->hit(r, 0.001f, INFINITY, rec))
        return background;

    ray scattered;
    color attenuation;
    color emitted = rec.material->emitted(rec.p);

    if (!rec.material->scatter(r, rec, attenuation, scattered, local_rand_state))
        return emitted;

    return emitted + attenuation * ray_color(scattered, background, world, local_rand_state, depth - 1);
}


__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    hittable_list* list = (hittable_list*)d_world;

    for (int i = 0; i < list->list_size; i++) {
        if (d_list[i]->material_ptr != NULL) {
            delete d_list[i]->material_ptr;
        }
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}



__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pixel, color background, camera** camera, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    color col = color(0.0f, 0.0f, 0.0f);
    int blocks = samples_per_pixel / TBP + 1;
    for (int k = 0; k < samples_per_pixel; k++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*camera)->get_ray(u, v, &local_rand_state);
        col += ray_color(r, background, world, &local_rand_state, 5);
    }
    col /= float(samples_per_pixel);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

int main() {
    //Image
   // const float aspect_ratio = 16.0f / 9.0f;
    const float aspect_ratio = 1;
    const int image_width = 600;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    int tx = 8;
    int ty = 8;
    int samples_per_pixel = 80;
    int num_of_spheres = 5;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    //Camera
    //on gpu

    //Render

    // allocate memory for pixels
    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(color);
    color* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    //Cuda Randomizer
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));


    scene* sc = new scene(2);
    //Start clock
    clock_t start, cuda_stop, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> > (fb, image_width, image_height, samples_per_pixel,sc->background_color, sc->cam->d_this, sc->world->d_this, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    cuda_stop = clock();
    double timer_seconds = ((double)(cuda_stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Cuda computation took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    uint8_t* pixels = new uint8_t[image_width * image_height * 3];
    int write_index = 0;
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            color pixelColor = fb[pixel_index];
            int ir = static_cast<int>(256 * clamp(pixelColor.r(), 0.0, 0.999));
            int ig = static_cast<int>(256 * clamp(pixelColor.g(), 0.0, 0.999));
            int ib = static_cast<int>(256 * clamp(pixelColor.b(), 0.0, 0.999));
            pixels[write_index++] = ir;
            pixels[write_index++] = ig;
            pixels[write_index++] = ib;
        }
    }

    stbi_write_bmp("image.bmp", image_width, image_height, 3, pixels);
    stop = clock();
    timer_seconds = ((double)(stop - cuda_stop)) / CLOCKS_PER_SEC;
    std::cerr << "Image write took " << timer_seconds << " seconds. \n";
 
    //Freeing memory
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (sc->world->d_list, sc->world->d_this, sc->cam->d_this);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    delete sc;


    hipDeviceReset();
    return 0;
}