#include "hip/hip_runtime.h"
﻿
#include "commons.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include<time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include "sphere.h"
#include "material.h"
#include "camera.h"
#include "hittable_list.h"
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__device__ color ray_color(const ray& r, const hittable** world, hiprandState* local_rand_state) {
    vec3 cur_attenuation = vec3(1.0f, 1.0f, 1.0f);
    ray cur_ray = r;
    for (int i = 0; i < 50; i++) {
        hit_record hit;
        if ((*world)->hit(cur_ray, 0.001f, INFINITY, hit)) {
            ray scattered;
            vec3 attenuation;
            if (hit.material->scatter(cur_ray, hit, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * color(1.0f, 1.0f, 1.0f) + t * color(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
    }
    // exceeded recursion value
    return color(0.0f, 0.0f, 0.0f);
}

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera, float aspect_ratio) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f, new lambertian(vec3(0.1f, 0.2f, 0.5f)));
        d_list[1] = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f, new lambertian(vec3(0.8f, 0.8f, 0.0f)));
        d_list[2] = new sphere(vec3(1.0f, 0.0f, -1.0f), 0.5f, new metal(vec3(0.8f, 0.6f, 0.2f), 1.0f));
        d_list[3] = new sphere(vec3(-1.0f, 0.0f, -1.0f), 0.5f, new dielectric(1.5f));
        d_list[4] = new sphere(vec3(-1.0f, 0.0f, -1.0f), -0.4f, new dielectric(1.5f));
        *d_world = new hittable_list(d_list, 5);
        vec3 lookfrom(3.0f, 3.0f, 2.0f);
        vec3 lookat(0.0f, 0.0f, -1.0f);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 2.0f;
        *d_camera = new camera(lookfrom,
            lookat,
            vec3(0.0f, 1.0f, 0.0f),
            20.0f,
            aspect_ratio,
            aperture,
            dist_to_focus);
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    hittable_list* list = (hittable_list*)d_world;

    for (int i = 0; i < list->list_size; i++) {
        delete ((sphere*)d_list[i])->material_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

__global__ void render(vec3* fb, int max_x, int max_y, int samples_per_pixel, camera** camera, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    color col = color(0.0f, 0.0f, 0.0f);
    for (int k = 0; k < samples_per_pixel; k++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*camera)->get_ray(u, v);
        col += ray_color(r, world, &local_rand_state);
    }
    col /= float(samples_per_pixel);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

int main() {
    //Image
    const float aspect_ratio = 16.0f / 9.0f;
    const int image_width = 800;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    int tx = 8;
    int ty = 8;
    int samples_per_pixel = 100;
    int num_of_spheres = 5;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    //Camera
    //on gpu

    //Render

    // allocate memory for pixels
    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(color);
    color* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    //Cuda Randomizer
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

    // allocate Memory for list of objects
    hittable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, num_of_spheres * sizeof(hittable*)));
    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world<<< 1, 1 >>>(d_list, d_world, d_camera, aspect_ratio);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());



    //Start clock
    clock_t start, cuda_stop, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> > (fb, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    cuda_stop = clock();
    double timer_seconds = ((double)(cuda_stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Cuda computation took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    uint8_t* pixels = new uint8_t[image_width * image_height * 3];
    int write_index = 0;
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            color pixelColor = fb[pixel_index];
            int ir = static_cast<int>(256 * clamp(pixelColor.r(), 0.0, 0.999));
            int ig = static_cast<int>(256 * clamp(pixelColor.g(), 0.0, 0.999));
            int ib = static_cast<int>(256 * clamp(pixelColor.b(), 0.0, 0.999));
            pixels[write_index++] = ir;
            pixels[write_index++] = ig;
            pixels[write_index++] = ib;
        }
    }

    stbi_write_bmp("image.bmp", image_width, image_height, 3, pixels);
    stop = clock();
    timer_seconds = ((double)(stop - cuda_stop)) / CLOCKS_PER_SEC;
    std::cerr << "Image write took " << timer_seconds << " seconds. \n";
 
    //Freeing memory
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
    return 0;
}